
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addStuff(int *d_a,int *d_b,int *d_c,int n)
{
	int i=blockIdx.x*blockDim.x+ threadIdx.x;

	if(i<n)
		d_c[i]=d_a[i]+d_b[i];

	printf("%d \n",d_c[i]);


}


int main()
{

	int n,size,i;
	int *a,*b,*c,*d_a,*d_b,*d_c;

	printf("Enter the nuber of elements: \n");
	scanf("%d",&n);

	a=(int *)malloc((size=sizeof(int)*n));
	b=(int *)malloc(size);
	c=(int *)malloc(size);


	hipMalloc((void **)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void **)&d_c,size);

	for(i=0;i<n;i++)
	{
		a[i]=i;
		b[i]=100;
	}

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	addStuff<<< (int)(n/1000.0 + 1),1000>>>(d_a,d_b,d_c,n);


	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);


	for(i=0;i<n;i++)
		printf("%d \n",c[i]);

	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);



	return 0;
}
